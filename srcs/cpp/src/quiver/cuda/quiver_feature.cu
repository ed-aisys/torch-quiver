#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <pybind11/numpy.h>
#include <quiver/common.hpp>
#include <quiver/quiver.cu.hpp>
#include <quiver/shard_tensor.cu.hpp>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

#include <torch/csrc/utils/python_numbers.h>
//#include <ATen/MapAllocator.h>
#include <atomic>
#include <string>

namespace quiver
{
#define CHECK_CPU(x)                                                          \
AT_ASSERTM(!x.device().is_cuda(), #x " must be CPU tensor")
class ShardTensorItem
{
  public:
    int device;
    std::string mem_handle;
    std::vector<int> shape;
    // for now we assume it is all float
    int dtype;
    ShardTensorItem(int device_, std::string mem_handle_, std::vector<int> shape_):device(device_), mem_handle(mem_handle_), shape(shape_)
    {

    }
    ShardTensorItem(){

    };
    void set_device(int device_){
        device = device;
    }
    void set_mem_handle(std::string mem_handle_){
        mem_handle = mem_handle_;
    }
    void set_shape(std::vector<int> shape_){
        shape = shape_;
    }

};

class ShardTensor
{
  public:
    ShardTensor(int device) : device_(device), inited_(false), device_count_(0)
    {
    }

    size_t get_tensor_bytes(torch::Tensor tensor){
        // assume it's float 
        int dim = tensor.dim();
        size_t total_bytes = 4;
        for(int index = 0; index < dim; index++){
            total_bytes *= tensor.sizes()[index];
        }
        return total_bytes;
    }
    std::vector<int> get_tensor_shape(torch::Tensor tensor){
        std::vector<int> shape; 
        int dim = tensor.dim();
        for(int index = 0; index < dim; index++){
            shape.push_back(tensor.sizes()[index]);
        }
        return shape;
    }

    void append(ShardTensorItem item){
        if (!inited_) {
            shape_.resize(item.shape.size());
            // std::cout<<"check shape_ size "<<shape_.size()<<std::endl;
            shape_[0] = 0;
            auto tensor_sizes = item.shape;
            for (int index = 1; index < shape_.size(); index++) {
                shape_[index] = tensor_sizes[index];
            }
            inited_ = true;
            offset_list_.push_back(0);
        }
        void *ptr = NULL;
        tensor_devices_.push_back(item.device);
        hipIpcOpenMemHandle(&ptr, *(hipIpcMemHandle_t *)item.mem_handle.data(), hipIpcMemLazyEnablePeerAccess);
        dev_ptrs_.push_back((float*)ptr);
        hipPointerAttribute_t attributes;
        hipPointerGetAttributes(&attributes, ptr);
        if(attributes.devicePointer == 0){
            printf("WARNING: Tensor from device %d can NOT be accessed in kernel launched on device %d \n", attributes.device, device_);
        }
        shape_[0] += item.shape[0];
        device_count_ += 1;
    }

    void append(torch::Tensor &tensor, int target_device)
    {
        CHECK_CPU(tensor);
        // for now, we assume tensor is added ordered
        if (!inited_) {
            shape_.resize(tensor.dim());
            // std::cout<<"check shape_ size "<<shape_.size()<<std::endl;
            shape_[0] = 0;
            auto tensor_sizes = tensor.sizes();
            for (int index = 1; index < shape_.size(); index++) {
                shape_[index] = tensor_sizes[index];
            }
            inited_ = true;
            offset_list_.push_back(0);
        }
        tensor_shapes_.push_back(get_tensor_shape(tensor));

        if (device_count_ > 0) {
            offset_list_.push_back(offset_list_[device_count_ - 1] +
                                   tensor.sizes()[0]);
        }
        void *ptr = NULL;
        size_t data_size = get_tensor_bytes(tensor);
        tensor_devices_.push_back(target_device);
        if(target_device >= 0){
            // if target_device >= 0, it means we use p2p 
            printf("LOG >>> Malloc Data On Device %d With %ulld Bytes\n", target_device, data_size);
            hipSetDevice(target_device);
            hipMalloc(&ptr, data_size);
            hipMemcpy(ptr, tensor.data_ptr<float>(), data_size, hipMemcpyHostToDevice);
            hipSetDevice(device_);
        }else{
            hipSetDevice(device_);
            // if target_device < 0, it means we use Zero-Copy 
            hipHostRegister(tensor.data_ptr<float>(), data_size, hipHostRegisterMapped);
            hipHostGetDevicePointer(&ptr, (void *)tensor.data_ptr<float>(), 0);
        }

        dev_ptrs_.push_back((float*)ptr);

        hipPointerAttribute_t attributes;
        hipPointerGetAttributes(&attributes, ptr);
        if(attributes.devicePointer == 0){
            printf("WARNING: Tensor from device %d can NOT be accessed in kernel launched on device %d \n", attributes.device, device_);
        }
        shape_[0] += tensor.size(0);
        device_count_ += 1;
    }


    torch::Tensor operator[](torch::Tensor &indices)
    {
        /*
        __global__ void quiver_tensor_gather(const int64_t** dev_ptrs, const
        int64_t* offsets, const int device_count, const int64_t* indices, int
        indice_length, const float* res, const int item_byte_size){
        torch::zeros((100,100),torch::KF32);
        */
        hipSetDevice(device_);
        auto stream = at::cuda::getCurrentCUDAStream();
        std::vector<int64_t> res_shape(shape_);
        res_shape[0] = indices.numel();
        // decide Tensor
        auto options = torch::TensorOptions()
                           .dtype(at::kFloat)
                           .device(torch::kCUDA, device_);
        auto res = torch::empty(res_shape, options);

        // Copy buffers Device
        float **buffers_device;
        hipMalloc((void ***)&buffers_device, sizeof(float *) * device_count_);
        hipMemcpy(buffers_device, &dev_ptrs_[0],
                   sizeof(float *) * dev_ptrs_.size(), hipMemcpyHostToDevice);
        cudaCheckError();
        // copy offset
        int64_t *offset_device;
        hipMalloc((void **)&offset_device,
                   sizeof(int64_t) * offset_list_.size());
        hipMemcpy(offset_device, &offset_list_[0],
                   sizeof(int64_t) * offset_list_.size(),
                   hipMemcpyHostToDevice);
        cudaCheckError();
        /*
        std::cout << "LOG >>> "
                  << " offset_size " << offset_list_.size() << " Offset Values "
                  << offset_list_[0] << ", " << offset_list_[1] << " stride "
                  << stride(0) << std::endl;
        */
        int blockSize = 0;
        int numBlocks = 0;
        hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize,
                                           quiver_tensor_gather);
        // std::cout<<"LOG >>> "<<" numBlocks "<< numBlocks <<" blockSize
        // "<<blockSize<<std::endl;

        quiver_tensor_gather<<<numBlocks, blockSize, 0, stream>>>(
            buffers_device, offset_device, offset_list_.size(),
            indices.data_ptr<int64_t>(), indices.numel(), res.data_ptr<float>(),
            stride(0));
        cudaCheckError();
        return res;
    }

    std::vector<int64_t> shape() const { return shape_; }

    int device() const { return device_; }

    int size(int dim) const { return shape_[dim]; }

    int64_t stride(int dim) const
    {
        int64_t res = 1;
        for (int index = dim + 1; index < shape_.size(); index++) {
            res *= shape_[index];
        }
        return res;
    }

    int64_t numel() const
    {
        int64_t res = 1;
        for (int index = 0; index < shape_.size(); index++) {
            res *= shape_[index];
        }
        return res;
    }
    std::vector<ShardTensorItem> share_ipc(){
        std::vector<ShardTensorItem> res;
        for(int index=0; index < dev_ptrs_.size(); index++){
            if(tensor_devices_[index] >= 0){
                hipIpcMemHandle_t handle;
                hipIpcGetMemHandle(&handle, dev_ptrs_[index]);
                void* ptr;
                hipIpcOpenMemHandle(&ptr, handle, hipIpcMemLazyEnablePeerAccess);
                hipPointerAttribute_t attributes;
                hipPointerGetAttributes(&attributes, ptr);
                printf("Tensor from device %d can be accessed in kernel launched on device %d by %d \n", attributes.device, device_, attributes.devicePointer);
                
                std::string string_handle((char *)&handle);
                ShardTensorItem item(tensor_devices_[index], string_handle, tensor_shapes_[index]);
                res.push_back(item);

            }
        }
        return res;
    }

    int device_count() const { return device_count_; }

  private:
    std::vector<int64_t> offset_list_;
    std::vector<float *> dev_ptrs_;
    std::vector<int> tensor_devices_;
    std::vector<std::vector<int>> tensor_shapes_;
    int device_;
    int device_count_;
    std::vector<int64_t> shape_;
    bool inited_;
};

void init_p2p(){
    std::cout << "LOG>>> P2P Access Initilization" << std::endl;
    int numGPUs;
    hipGetDeviceCount(&numGPUs);
    for (int i = 0; i < numGPUs; i++) {
        hipSetDevice(i);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        // CUDA IPC is only supported on devices with unified addressing
        if (!prop.unifiedAddressing) {
            printf("Device %d does not support unified addressing, skipping...\n", i);
            continue;
        }
        // This sample requires two processes accessing each device, so we need
        // to ensure exclusive or prohibited mode is not set
        if (prop.computeMode != hipComputeModeDefault) {
            printf("Device %d is in an unsupported compute mode for this sample\n",
                i);
            continue;
        }
        
        for (int j = i + 1; j < numGPUs; j++) {
            int access_i_j = 0;
            int access_j_i = 0;
            printf("Enable P2P Access Between %d ---> %d \n", i, j);
            hipDeviceCanAccessPeer(&access_i_j, i, j);
            hipDeviceCanAccessPeer(&access_j_i, j, i);
            if (access_i_j && access_j_i) {
                hipSetDevice(i);
                hipDeviceEnablePeerAccess(j, 0);
                cudaCheckError();
                hipSetDevice(j);
                hipDeviceEnablePeerAccess(i, 0);
                cudaCheckError();
            }
        }
    }
}
}  // namespace quiver
void register_cuda_quiver_feature(pybind11::module &m)
{
    m.def("init_p2p", &quiver::init_p2p,
            py::call_guard<py::gil_scoped_release>());
    
    
    py::class_<quiver::ShardTensorItem>(m, "ShardTensorItem")
        .def(py::init<>())
        .def_readwrite("device", &quiver::ShardTensorItem::device)
        .def_readwrite("shape", &quiver::ShardTensorItem::shape)
        .def_readwrite("mem_handle", &quiver::ShardTensorItem::mem_handle);
    

    py::class_<quiver::ShardTensor>(m, "ShardTensor")
        //.def(py::init<std::vector<torch::Tensor>, int>())
        .def(py::init<int>())
        .def("__getitem__", &quiver::ShardTensor::operator[],
             py::call_guard<py::gil_scoped_release>())
        .def("shape", &quiver::ShardTensor::shape,
             py::call_guard<py::gil_scoped_release>())
        .def("numel", &quiver::ShardTensor::numel,
             py::call_guard<py::gil_scoped_release>())
        .def("device", &quiver::ShardTensor::device,
             py::call_guard<py::gil_scoped_release>())
        .def("stride", &quiver::ShardTensor::stride,
             py::call_guard<py::gil_scoped_release>())
        .def("size", &quiver::ShardTensor::size,
             py::call_guard<py::gil_scoped_release>())
        .def("device_count", &quiver::ShardTensor::device_count,
             py::call_guard<py::gil_scoped_release>())
        .def("append", py::overload_cast<torch::Tensor&, int>(&quiver::ShardTensor::append),
             py::call_guard<py::gil_scoped_release>())
        .def("append", py::overload_cast<quiver::ShardTensorItem>(&quiver::ShardTensor::append),
             py::call_guard<py::gil_scoped_release>())
        .def("share_ipc", &quiver::ShardTensor::share_ipc,
             py::call_guard<py::gil_scoped_release>());
            
}
